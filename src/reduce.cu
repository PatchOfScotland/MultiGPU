#include "hip/hip_runtime.h"
#include <functional>
#include <sys/time.h>

#include "reduce/cpu.h"
#include "reduce/multiGPU.h"
#include "reduce/singleGPU.h"
#include "shared_cuda.cu.h"
#include "shared.h"

typedef float array_type;
typedef double return_type;

template<typename I, typename R>
class Add {
    public:
        typedef I InputElement;
        typedef R ReturnElement;

        static __device__ __host__ ReturnElement apply(
            const InputElement i, const ReturnElement r
        ) {
            return i+r;
        };
};


int main(int argc, char** argv){
    if (argc < 3)
    {
        std::cout << "Usage: " 
                  << argv[0] 
                  << " <array length> <benchmark repeats> -v(optional) -s(optional)\n";
        exit(EXIT_FAILURE);
    } 

    unsigned long int array_len = strtoul(argv[1], NULL, 0);
    unsigned int runs = atoi(argv[2]);
    bool validating = false;
    bool skip = false;

    for (int i=0; i<argc; i++) {
        if (strcmp(argv[i], "-v") == 0) {
            validating = true;
        }
        if (strcmp(argv[i], "-s") == 0) {
            skip = true;
        }

    }

    double datasize = ((array_len*sizeof(array_type))/1e9);
    std::cout << "Running array of length " 
              << array_len 
              << " (" 
              << datasize 
              <<"GB)\n";
    if (validating) {
        std::cout << "Will validate output\n";
    }
    else {
        std::cout << "Skipping output validation\n";
    }
    if (skip) {
        std::cout << "Skipping any significant processing\n";
    }

    array_type* input_array;
    return_type* output;
    return_type validation_result;
    hipEvent_t start_event;
    hipEvent_t end_event;
    float runtime_ms;
    float cpu_time_ms = -1;
    float single_gpu_time_ms = -1;
    float multi_gpu_time_ms = -1;

    CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
    CCC(hipMallocManaged(&output, sizeof(return_type)));

    CCC(hipEventCreate(&start_event));
    CCC(hipEventCreate(&end_event));
    float* timing_ms = (float*)calloc(runs, sizeof(float));

    int origin_device;
    CCC(hipGetDevice(&origin_device));
    int device_count;
    CCC(hipGetDeviceCount(&device_count));

    std::cout << "Initialising input array\n";
    if (skip == false) {
        init_array(input_array, array_len);
    }

    if (validating) { // Populate validation array
        std::cout << "Getting CPU result for validation\n";

        struct timeval cpu_start_time;
        struct timeval cpu_end_time;

        gettimeofday(&cpu_start_time, NULL);

        if (skip == false) {
            cpuReduction(
                reduction<array_type,return_type>, input_array, 
                &validation_result, array_len
            );    
        }
        gettimeofday(&cpu_end_time, NULL); 

        cpu_time_ms = (cpu_end_time.tv_usec+(1e6*cpu_end_time.tv_sec)) 
            - (cpu_start_time.tv_usec+(1e6*cpu_start_time.tv_sec));
        std::cout << "CPU reduction took: " << cpu_time_ms << "ms\n";
    }

    check_device_count();

    { // Benchmark a single GPU
        std::cout << "\nBenchmarking single GPU reduce ********************\n";

        std::cout << "  Running a warmup\n";
        singleGpuReduction<Add<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        for (int run=0; run<runs; run++) {
            CCC(hipEventRecord(start_event));
            singleGpuReduction<Add<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            print_loop_feedback(run, runs);

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
        }

         single_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    { // Benchmark multi GPU
        std::cout << "\nBenchmarking multi GPU reduce *********************\n";

        unsigned long int per_device = array_len / device_count;
        int remainder = array_len % device_count;
        unsigned long int running_total = 0;
        unsigned long int device_start;
        unsigned long int this_block;
        for (int device=0; device<device_count; device++) {           
            device_start = running_total;
            this_block = (remainder > 0) ? per_device + 1 : per_device;
            remainder -= 1;
            running_total += this_block;
            
            std::cout << "  A:" << input_array+device_start << "\n";
            std::cout << "  B:" << this_block*sizeof(array_type) << "\n";
            std::cout << "  B.5:" << this_block << "\n";
            std::cout << "  C:" << hipMemAdviseSetPreferredLocation << "\n";
            std::cout << "  D:" << device << "\n";

            CCC(hipMemAdvise(
                input_array+device_start, 
                this_block*sizeof(array_type), 
                hipMemAdviseSetPreferredLocation, 
                device
            ));
        }

        std::cout << "  Running a warmup\n";
        multiGpuReduction<Add<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));


        for (int run=0; run<runs; run++) {
            CCC(hipEventRecord(start_event));
            multiGpuReduction<Add<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            print_loop_feedback(run, runs);

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
        }

         multi_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    { // Benchmark multi GPU with hints
        std::cout << "\nBenchmarking multi GPU reduce with hints **********\n";

        std::cout << "  Running a warmup\n";
        multiGpuReduction<Add<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        for (int run=0; run<runs; run++) {
            CCC(hipEventRecord(start_event));
            multiGpuReduction<Add<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            print_loop_feedback(run, runs);

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
        }

         multi_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    hipFree(input_array);
    hipFree(output);
}
