#include <functional>
#include <sys/time.h>

#include "reduce/cpu.h"
#include "reduce/multiGPU.h"
#include "reduce/multiGPUstreams.h"
#include "reduce/singleGPU.h"
#include "shared.cu.h"
#include "shared.h"

typedef float array_type;

int main(int argc, char** argv){
    if (argc < 3)
    {
        std::cout << "Usage: " 
                  << argv[0] 
                  << " <array length> <benchmark repeats> -v(optional)\n";
        exit(EXIT_FAILURE);
    } 

    unsigned long int array_len = strtoul(argv[1], NULL, 0);
    unsigned int runs = atoi(argv[2]);
    bool validating = false;

    for (int i=0; i<argc; i++) {
        if (strcmp(argv[i], "-v") == 0) {
            validating = true;
        }
    }

    double datasize = ((array_len*sizeof(array_type))/1e9);
    std::cout << "Running array of length " 
              << array_len 
              << " (" 
              << datasize 
              <<"GB)\n";
    if (validating) {
        std::cout << "Will validate output\n";
    }
    else {
        std::cout << "Skipping output validation\n";
    }

    array_type* input_array;
    array_type* output;
    array_type validation_result;
    array_type constant = 0.1;
    hipEvent_t start_event;
    hipEvent_t end_event;
    float runtime_ms;
    long int cpu_time_ms = -1;
    long int single_gpu_time_ms = -1;
    long int multi_gpu_time_ms = -1;

    CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
    CCC(hipMallocManaged(&output, sizeof(array_type)));

    CCC(hipEventCreate(&start_event));
    CCC(hipEventCreate(&end_event));
    float* timing_ms = (float*)calloc(runs, sizeof(float));

    int origin_device;
    CCC(hipGetDevice(&origin_device));
    int device_count;
    CCC(hipGetDeviceCount(&device_count));

    std::cout << "Initialising input array\n";
    init_array(input_array, array_len);

    if (validating) { // Populate validation array
        std::cout << "Getting CPU result for validation\n";

        struct timeval cpu_start_time;
        struct timeval cpu_end_time;

        gettimeofday(&cpu_start_time, NULL);

        cpuReduction(
            reduction<array_type>, input_array, &validation_result, array_len
        );    
        gettimeofday(&cpu_end_time, NULL); 

        cpu_time_ms = (cpu_end_time.tv_usec+(1e6*cpu_end_time.tv_sec)) 
            - (cpu_start_time.tv_usec+(1e6*cpu_start_time.tv_sec));
        std::cout << "CPU reduction took: " << cpu_time_ms << "ms\n";
    }

    std::cout << "Validation result is: " << validation_result << "\n";

    hipFree(input_array);
    hipFree(output);
}
