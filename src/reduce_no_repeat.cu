#include "hip/hip_runtime.h"
#include <functional>
#include <sys/time.h>

#include "reduce/cpu.h"
#include "reduce/multiGPU.h"
#include "reduce/singleGPU.h"
#include "shared_cuda.cu.h"
#include "shared.h"

typedef float array_type;
typedef double return_type;

template<typename I, typename R>
class Add {
    public:
        typedef I InputElement;
        typedef R ReturnElement;
        static const bool commutative = true;

        static __device__ __host__ ReturnElement apply(
            const InputElement i, const ReturnElement r
        ) {
            return i+r;
        };

        static __device__ __host__ ReturnElement init () {
            return (ReturnElement)0;
        }

        static __device__ __host__ ReturnElement map (
            const InputElement &i
        ) {
            return (ReturnElement)i;
        }

        static __device__ __host__ ReturnElement remVolatile (
            volatile ReturnElement &i
        ) {
            return i;
        }
};

template<typename I, typename R>
class AddNonCommutative {
    public:
        typedef I InputElement;
        typedef R ReturnElement;
        static const bool commutative = false;

        static __device__ __host__ ReturnElement apply(
            const InputElement i, const ReturnElement r
        ) {
            return i+r;
        };

        static __device__ __host__ ReturnElement init () {
            return (ReturnElement)0;
        }

        static __device__ __host__ ReturnElement map (
            const InputElement &i
        ) {
            return (ReturnElement)i;
        }

        static __device__ __host__ ReturnElement remVolatile (
            volatile ReturnElement &i
        ) {
            return i;
        }
};


int main(int argc, char** argv){
    if (argc < 3)
    {
        std::cout << "Usage: " 
                  << argv[0] 
                  << " <array length> <benchmark repeats> -v(optional) -s(optional) -r(optional)\n";
        exit(EXIT_FAILURE);
    } 

    unsigned long int array_len = strtoul(argv[1], NULL, 0);
    unsigned int runs = atoi(argv[2]);
    bool validating = false;
    bool skip = false;
    bool reduced_output = false;

    for (int i=0; i<argc; i++) {
        if (strcmp(argv[i], "-v") == 0) {
            validating = true;
        }
        if (strcmp(argv[i], "-s") == 0) {
            skip = true;
        }
        if (strcmp(argv[i], "-r") == 0) {
            reduced_output = true;
        }
    }

    double datasize = ((array_len*sizeof(array_type))/1e9);
    std::cout << "Running array of length " 
              << array_len 
              << " (" 
              << datasize 
              <<"GB)\n";
    if (validating) {
        std::cout << "Will validate output\n";
    }
    else {
        std::cout << "Skipping output validation\n";
    }
    if (skip) {
        std::cout << "Skipping any significant processing\n";
    }

    array_type* input_array;
    return_type* output;
    return_type validation_result;
    hipEvent_t start_event;
    hipEvent_t end_event;
    float runtime_ms;
    float cpu_time_ms = -1;
    float single_gpu_time_ms = -1;
    float multi_gpu_time_ms = -1;


    CCC(hipEventCreate(&start_event));
    CCC(hipEventCreate(&end_event));
    float* timing_ms = (float*)calloc(runs, sizeof(float));

    int origin_device;
    CCC(hipGetDevice(&origin_device));
    int device_count;
    CCC(hipGetDeviceCount(&device_count));

    check_device_count();

    { // Get CPU baseline
        std::cout << "Getting CPU result\n";

        CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
        CCC(hipMallocManaged(&output, sizeof(return_type)));
        init_sparse_array(input_array, array_len, 10000);

        struct timeval cpu_start_time;
        struct timeval cpu_end_time;

        gettimeofday(&cpu_start_time, NULL);

        if (skip == false) {
            cpuReduction<Add<array_type,return_type>>(
                input_array, 
                &validation_result, array_len
            );    
        }
        gettimeofday(&cpu_end_time, NULL); 

        CCC(hipFree(input_array));
        CCC(hipFree(output));

        cpu_time_ms = (cpu_end_time.tv_usec+(1e6*cpu_end_time.tv_sec)) 
            - (cpu_start_time.tv_usec+(1e6*cpu_start_time.tv_sec));
        std::cout << "CPU reduction took: " << cpu_time_ms << "ms\n";
        std::cout << "CPU throughput:     " << (float)datasize / cpu_time_ms << "GB/sec\n";
    }

    { // Benchmark commutative single GPU
        std::cout << "\nBenchmarking commutative single GPU reduce ********\n";

        std::cout << "  Running a warmup\n";

        CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
        CCC(hipMallocManaged(&output, sizeof(return_type)));
        init_sparse_array(input_array, array_len, 10000);

        singleGpuReduction<Add<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        CCC(hipFree(input_array));
        CCC(hipFree(output));

        for (int run=0; run<runs; run++) {
            CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
            CCC(hipMallocManaged(&output, sizeof(return_type)));
            init_sparse_array(input_array, array_len, 10000);

            CCC(hipEventRecord(start_event));
            singleGpuReduction<Add<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            if (reduced_output == false) {
                print_loop_feedback(run, runs);
            }

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
            CCC(hipFree(input_array));
            CCC(hipFree(output));
        }

         single_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    { // Benchmark commutative multi GPU
        std::cout << "\nBenchmarking commutative multi GPU reduce *********\n";

        CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
        CCC(hipMallocManaged(&output, sizeof(return_type)));
        init_sparse_array(input_array, array_len, 10000);

        std::cout << "  Running a warmup\n";
        multiGpuReduction<Add<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        CCC(hipFree(input_array));
        CCC(hipFree(output));

        for (int run=0; run<runs; run++) {
            CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
            CCC(hipMallocManaged(&output, sizeof(return_type)));
            init_sparse_array(input_array, array_len, 10000);

            CCC(hipEventRecord(start_event));
            multiGpuReduction<Add<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            if (reduced_output == false) {
                print_loop_feedback(run, runs);
            }

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
            CCC(hipFree(input_array));
            CCC(hipFree(output));
        }

         multi_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    { // Benchmark commutative multi GPU with hints
        std::cout << "\nBenchmarking commutative multi GPU reduce with hints\n";
        CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
        CCC(hipMallocManaged(&output, sizeof(return_type)));
        init_sparse_array(input_array, array_len, 10000);

        unsigned long int per_device = array_len / device_count;
        int remainder = array_len % device_count;
        unsigned long int running_total = 0;
        unsigned long int device_start;
        unsigned long int this_block;
        for (int device=0; device<device_count; device++) {           
            device_start = running_total;
            this_block = (remainder > 0) ? per_device + 1 : per_device;
            remainder -= 1;
            running_total += this_block;

            CCC(hipMemAdvise(
                input_array+device_start, 
                this_block*sizeof(array_type), 
                hipMemAdviseSetPreferredLocation, 
                device
            ));
        }

        std::cout << "  Running a warmup\n";
        multiGpuReduction<Add<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        CCC(hipFree(input_array));
        CCC(hipFree(output));

        for (int run=0; run<runs; run++) {
            CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
            CCC(hipMallocManaged(&output, sizeof(return_type)));
            init_sparse_array(input_array, array_len, 10000);
            
            for (int device=0; device<device_count; device++) {           
                device_start = running_total;
                this_block = (remainder > 0) ? per_device + 1 : per_device;
                remainder -= 1;
                running_total += this_block;

                CCC(hipMemAdvise(
                    input_array+device_start, 
                    this_block*sizeof(array_type), 
                    hipMemAdviseSetPreferredLocation, 
                    device
                ));
            }

            CCC(hipEventRecord(start_event));
            multiGpuReduction<Add<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            if (reduced_output == false) {
                print_loop_feedback(run, runs);
            }

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
            CCC(hipFree(input_array));
            CCC(hipFree(output));
        }

         multi_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    single_gpu_time_ms = -1;
    multi_gpu_time_ms = -1;

    { // Benchmark associative single GPU
        std::cout << "\nBenchmarking associative single GPU reduce ****\n";
        CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
        CCC(hipMallocManaged(&output, sizeof(return_type)));
        init_sparse_array(input_array, array_len, 10000);

        std::cout << "  Running a warmup\n";
        singleGpuReduction<AddNonCommutative<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        CCC(hipFree(input_array));
        CCC(hipFree(output));

        for (int run=0; run<runs; run++) {
            CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
            CCC(hipMallocManaged(&output, sizeof(return_type)));
            init_sparse_array(input_array, array_len, 10000);

            CCC(hipEventRecord(start_event));
            singleGpuReduction<AddNonCommutative<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            if (reduced_output == false) {
                print_loop_feedback(run, runs);
            }

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
            CCC(hipFree(input_array));
            CCC(hipFree(output));
        }

         single_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    { // Benchmark associative multi GPU
        std::cout << "\nBenchmarking associative multi GPU reduce *****\n";
        CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
        CCC(hipMallocManaged(&output, sizeof(return_type)));
        init_sparse_array(input_array, array_len, 10000);

        std::cout << "  Running a warmup\n";
        multiGpuReduction<AddNonCommutative<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        CCC(hipFree(input_array));
        CCC(hipFree(output));

        for (int run=0; run<runs; run++) {
            CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
            CCC(hipMallocManaged(&output, sizeof(return_type)));
            init_sparse_array(input_array, array_len, 10000);

            CCC(hipEventRecord(start_event));
            multiGpuReduction<AddNonCommutative<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            if (reduced_output == false) {
                print_loop_feedback(run, runs);
            }

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
            CCC(hipFree(input_array));
            CCC(hipFree(output));
        }

         multi_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

    { // Benchmark associative multi GPU with hints
        std::cout << "\nBenchmarking associative multi GPU reduce with hints\n";
        CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
        CCC(hipMallocManaged(&output, sizeof(return_type)));
        init_sparse_array(input_array, array_len, 10000);

        unsigned long int per_device = array_len / device_count;
        int remainder = array_len % device_count;
        unsigned long int running_total = 0;
        unsigned long int device_start;
        unsigned long int this_block;
        for (int device=0; device<device_count; device++) {           
            device_start = running_total;
            this_block = (remainder > 0) ? per_device + 1 : per_device;
            remainder -= 1;
            running_total += this_block;

            CCC(hipMemAdvise(
                input_array+device_start, 
                this_block*sizeof(array_type), 
                hipMemAdviseSetPreferredLocation, 
                device
            ));
        }

        std::cout << "  Running a warmup\n";
        multiGpuReduction<AddNonCommutative<array_type,return_type>>(
            input_array, output, array_len, skip
        );
        CCC(hipEventRecord(end_event));
        CCC(hipEventSynchronize(end_event));

        CCC(hipFree(input_array));
        CCC(hipFree(output));

        for (int run=0; run<runs; run++) {
            CCC(hipMallocManaged(&input_array, array_len*sizeof(array_type)));
            CCC(hipMallocManaged(&output, sizeof(return_type)));
            init_sparse_array(input_array, array_len, 10000);

            for (int device=0; device<device_count; device++) {           
                device_start = running_total;
                this_block = (remainder > 0) ? per_device + 1 : per_device;
                remainder -= 1;
                running_total += this_block;

                CCC(hipMemAdvise(
                    input_array+device_start, 
                    this_block*sizeof(array_type), 
                    hipMemAdviseSetPreferredLocation, 
                    device
                ));
            }

            CCC(hipEventRecord(start_event));
            multiGpuReduction<AddNonCommutative<array_type,return_type>>(
                input_array, output, array_len, skip
            );
            CCC(hipEventRecord(end_event));
            CCC(hipEventSynchronize(end_event));
            CCC(hipPeekAtLastError());

            CCC(hipEventElapsedTime(&runtime_ms, start_event, end_event));
            timing_ms[run] = runtime_ms;

            if (reduced_output == false) {
                print_loop_feedback(run, runs);
            }

            // do this at the end as reading output array will shift it back to 
            // the host
            if (validating && run==runs-1) {
                array_type tolerance = array_len / 1e5;
                std::cout << "  Comparing " 
                          << std::setprecision(12) 
                          << validation_result 
                          << " and " 
                          << std::setprecision(12) 
                          << *output 
                          << " with tolerance of " 
                          << tolerance 
                          << "\n";
                // Very much rough guess
                if (in_range<double>(validation_result, *output, tolerance)) {
                    std::cout << "  Result is correct\n";
                } else {
                    std::cout << "  Result is incorrect. Skipping any "
                              << "subsequent runs\n";
                    break;
                }
            }
            CCC(hipFree(input_array));
            CCC(hipFree(output));
        }

         multi_gpu_time_ms = print_timing_stats(
            timing_ms, runs, datasize, cpu_time_ms, single_gpu_time_ms, 
            multi_gpu_time_ms
        );
    }

}
